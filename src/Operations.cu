/*
 *	The MIT License (MIT)
 *	Copyright (c) 2013-2015 Wei Dai
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in
 *	all copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *	THE SOFTWARE.
 */
#include "Operations.h"
#include "Parameters.h"
#include "DeviceManager.h"
#include "Debug.h"
#include "Base.h"
#include "CuHE.h"

namespace cuHE {
///////////////////////////////////////////////////////////////////////////////
//// Pre-computation //////////////////////////////////////////////////////////
static ZZ* crtPrime; // decreasing?
static ZZ* coeffModulus; // decreasing
void genCrtPrimes() {
	int pnum = param.numCrtPrime;
	crtPrime = new ZZ[pnum];
	unsigned* h_p = new unsigned[pnum];
	int logmid = param.logCoeffMin-(pnum-param.depth)*param.logCrtPrime;
	// after cutting, fairly larger primes
	ZZ temp = to_ZZ(0x1<<param.logCrtPrime)-1;
	for (int i=0; i<=pnum-param.depth-1; i++) {
		while (!ProbPrime(temp, 10))
			temp --;
		conv(h_p[i], temp);
		crtPrime[i] = temp;
		temp --;
	}

	// mid
	ZZ tmid;
	if (logmid != param.logCrtPrime)
		tmid = to_ZZ(0x1<<logmid)-1;
	else
		tmid = temp;
	while (!ProbPrime(tmid, 10))
		tmid --;
	conv(h_p[pnum-param.depth], tmid);
	crtPrime[pnum-param.depth] = tmid;

	// for cutting
	if (param.logCoeffCut == logmid)
		temp = tmid-1;
	else if (param.logCoeffCut == param.logCrtPrime)
		temp --;
	else
		temp = to_ZZ(0x1<<param.logCoeffCut)-1;
	for (int i=pnum-param.depth+1; i<pnum; i++) {
		while (!ProbPrime(temp, 10) || temp%to_ZZ(param.modMsg) != 1)
			temp --;
		conv(h_p[i], temp);
		crtPrime[i] = temp;
		temp --;
	}

	preload_crt_p(h_p, pnum);
	delete [] h_p;
};
void genCoeffModuli() {
	int d = param.depth;
	int pnum = param.numCrtPrime;
	coeffModulus = new ZZ[d];
	for (int i=0; i<d; i++) {
		coeffModulus[i] = 1;
		for (int j=0; j<pnum-i; j++)
			coeffModulus[i] *= crtPrime[j];
	}
}
void genCrtInvPrimes() {
	int pnum = param.numCrtPrime;
	uint32 *h_pinv = new uint32[pnum*(pnum-1)/2];
	ZZ temp;
	for (int i=1; i<pnum; i++)
		for (int j=0; j<i; j++)
			conv(h_pinv[i*(i-1)/2+j], InvMod(crtPrime[i]%crtPrime[j], crtPrime[j]));
	preload_crt_invp(h_pinv, pnum*(pnum-1)/2);
	delete [] h_pinv;
}
static int* icrtLevel; // one int for each device
static struct IcrtConst {
	uint32 *q;
	uint32 *qp;
	uint32 *qpinv;
} **icrtConst;
void genIcrtByLevel(int lvl) {
	int pnum = param._numCrtPrime(lvl);
	int words_q = param._wordsCoeff(lvl);
	int words_qp = param._wordsCoeff(lvl+1);
	for (int dev=0; dev<numDevices(); dev++) {
		CSC(hipSetDevice(dev));
		CSC(hipHostMalloc(&icrtConst[dev][lvl].q,
				words_q*sizeof(uint32)));
		CSC(hipHostMalloc(&icrtConst[dev][lvl].qp,
				pnum*words_qp*sizeof(uint32)));
		CSC(hipHostMalloc(&icrtConst[dev][lvl].qpinv,
				pnum*sizeof(uint32)));
	}
	ZZ *z_qp = new ZZ[pnum];
	for (int i=0; i<pnum; i++)
		z_qp[i] = coeffModulus[lvl]/crtPrime[i];
	for (int dev=0; dev<numDevices(); dev++) {
		BytesFromZZ((uint8 *)icrtConst[dev][lvl].q,
				coeffModulus[lvl], words_q*sizeof(uint32));
		for (int i=0; i<pnum; i++) {
			BytesFromZZ((uint8 *)(&icrtConst[dev][lvl].qp[words_qp*i]),
					z_qp[i], words_qp*sizeof(uint32));
			conv(icrtConst[dev][lvl].qpinv[i],
					InvMod(z_qp[i]%crtPrime[i], crtPrime[i]));
		}
	}
	delete [] z_qp;
};
void genIcrt() {
	icrtConst = new IcrtConst *[numDevices()];
	icrtLevel = new int[numDevices()];
	for (int dev=0; dev<numDevices(); dev++) {
		icrtConst[dev] = new IcrtConst[param.depth];
		icrtLevel[dev] = -1;
	}
	for (int i=0; i<param.depth; i++)
		genIcrtByLevel(i);
};
void loadIcrtConst(int lvl, int dev, hipStream_t st) {
	if (icrtLevel[dev] != lvl) {
		int pnum = param._numCrtPrime(lvl);
		int words_q = param._wordsCoeff(lvl);
		int words_qp = param._wordsCoeff(lvl+1);
		CSC(hipSetDevice(dev));
		load_icrt_M(icrtConst[dev][lvl].q, words_q, dev, st);
		load_icrt_mi(icrtConst[dev][lvl].qp, words_qp*pnum, dev, st);
		load_icrt_bi(icrtConst[dev][lvl].qpinv, pnum, dev, st);
		icrtLevel[dev] = lvl;
	}
};
void getCoeffModuli(ZZ* dst) {
	for (int i=0; i<param.depth; i++)
		dst[i] = coeffModulus[i];
}
void initCrt(ZZ* coeffModulus) {
	genCrtPrimes();
	genCoeffModuli();
	genCrtInvPrimes();
	genIcrt();
	for (int dev=0; dev<numDevices(); dev++)
		loadIcrtConst(0, dev);
	getCoeffModuli(coeffModulus);
}
///////////////////////////////////////////////////////////////////////////////
static uint64 **d_swap; // conversion buffer
static uint32 **d_hold; // intt result buffer
void initNtt() {
	// twiddle factors
	const ZZ P = to_ZZ(0xffffffff00000001);
	const ZZ g = to_ZZ((uint64)15893793146607301539);
	int e0 = 65536/param.nttLen;
	ZZ w0 =	PowerMod(g, e0, P);
	uint64 *h_roots = new uint64[param.nttLen];
	for (int i=0; i<param.nttLen; i++)
		conv(h_roots[i], PowerMod(w0, i, P));
	preload_ntt(h_roots, param.nttLen);
	delete [] h_roots;
	// temporary result allocation
	d_swap = new uint64 *[numDevices()];
	d_hold = new uint32 *[numDevices()];
	for (int dev=0; dev<numDevices(); dev++) {
		hipSetDevice(dev);
		CSC(hipMalloc(&d_swap[dev], param.nttLen*sizeof(uint64)));
		CSC(hipMalloc(&d_hold[dev], param.numCrtPrime*param.nttLen*sizeof(uint32)));
	}
}
uint32 *inttResult(int dev) {
	return ptrNttHold(dev);
}
uint64 **ptrNttSwap() { return d_swap;}
uint32 **ptrNttHold() {	return d_hold;}
uint64 *ptrNttSwap(int dev) { return d_swap[dev];}
uint32 *ptrNttHold(int dev) { return d_hold[dev];}
///////////////////////////////////////////////////////////////////////////////
uint64 **d_barrett_ntt;
uint32 **d_barrett_crt;
uint32 **d_barrett_src;
void createBarrettTemporySpace() {
	d_barrett_crt = new uint32*[numDevices()];
	d_barrett_ntt = new uint64*[numDevices()];
	d_barrett_src = new uint32*[numDevices()];
	for (int dev=0; dev<numDevices(); dev++) {
		hipSetDevice(dev);
		CSC(hipMalloc(&d_barrett_crt[dev], param.numCrtPrime*param.nttLen*sizeof(uint32)));
		CSC(hipMalloc(&d_barrett_ntt[dev], param.numCrtPrime*param.nttLen*sizeof(uint64)));
		CSC(hipMalloc(&d_barrett_src[dev], param.numCrtPrime*param.nttLen*sizeof(uint32)));
	}
}
static uint32 *ptrBarrettCrt(int dev) { return d_barrett_crt[dev];}
static uint64 *ptrBarrettNtt(int dev) { return d_barrett_ntt[dev];}
static uint32 *ptrBarrettSrc(int dev) { return d_barrett_src[dev];}
void setPolyModulus(ZZX m) {
	// compute NTL type zm, zu
	ZZ zq = coeffModulus[0];
	ZZX zm = m;
	ZZX zu;
	SetCoeff(zu, 2*param.modLen-1, 1);
	zu /= zm;
	for (int i=0; i<=deg(zm); i++)
		SetCoeff(zm, i, coeff(zm, i)%zq);
	for (int i=0; i<=deg(zu); i++)
		SetCoeff(zu, i, coeff(zu, i)%zq);
	SetCoeff(zm, param.modLen, 0);
	// prep m
	CuCtxt c;
	c.setLevel(0, 0, zm);
	c.x2c();
	preload_barrett_m_c(c.cRep(), param.numCrtPrime*param.crtLen*sizeof(uint32));
	c.x2n();
	preload_barrett_m_n(c.nRep(), param.numCrtPrime*param.nttLen*sizeof(uint64));
	c.~CuCtxt();	
	// prep u
	CuCtxt cc;
	cc.setLevel(0, 0, zu);
	cc.x2n();
	preload_barrett_u_n(cc.nRep(), param.numCrtPrime*param.nttLen*sizeof(uint64));
	cc.~CuCtxt();
};
void initBarrett(ZZX m) {
	setPolyModulus(m);
	createBarrettTemporySpace();
}

///////////////////////////////////////////////////////////////////////////////
//// Operations ///////////////////////////////////////////////////////////////
void crt(uint32 *dst, uint32 *src, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	hipSetDevice(dev);
	crt<<<(param.modLen+63)/64, 64, param._wordsCoeff(lvl)*sizeof(uint32)*64, st>>>
			(dst, src, param._numCrtPrime(lvl), param._wordsCoeff(lvl), param.modLen, param.crtLen);
	CCE();
}
void icrt(uint32 *dst, uint32 *src, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	loadIcrtConst(lvl, dev, st);
	CSC(hipStreamSynchronize(st));
	CSC(hipSetDevice(dev));
	icrt<<<(param.modLen+63)/64, 64, 0, st>>>(dst, src, param._numCrtPrime(lvl),
			param._wordsCoeff(lvl), param._wordsCoeff(lvl+1), param.modLen, param.crtLen);
	CCE();
}
void crtAdd(uint32 *sum, uint32 *x, uint32 *y, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	hipSetDevice(dev);
	crt_add<<<(param.modLen+63)/64, 64, 0, st>>>(sum, x, y, param._numCrtPrime(lvl), param.modLen, param.crtLen);
	CCE();
}
void crtAddInt(uint32 *sum, uint32 *x, unsigned a, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	hipSetDevice(dev);
	crt_add_int<<<(param._numCrtPrime(lvl)+63)/64, 64, 0, st>>>(sum, x, a, param._numCrtPrime(lvl), param.crtLen);
	CCE();
}
void crtAddNX1(uint32 *sum, uint32 *x, uint32 *scalar, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	hipSetDevice(dev);
	crt_add_nx1<<<(param.modLen+63)/64, 64, 0, st>>>(sum, x, scalar, param._numCrtPrime(lvl), param.modLen, param.crtLen);
	CCE();
}
void crtMulInt(uint32 *prod, uint32 *x, int a, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	hipSetDevice(dev);
	crt_mul_int<<<(param.numCrtPrime-lvl+63)/64, 64, 0, st>>>(prod, x, a, param._numCrtPrime(lvl), param.crtLen);
	CCE();
}
void crtModSwitch(uint32 *dst, uint32 *src, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	hipSetDevice(dev);
	modswitch<<<(param.modLen+63)/64, 64, 0, st>>>(dst, src, param._numCrtPrime(lvl),
			param.modLen, param.crtLen, param.modMsg);
	CCE();
}

//// single crt polynomial
void _ntt(uint64 *X, uint32 *x, int dev, hipStream_t st) {
	if (param.nttLen == 16384) {
		ntt_1_16k_ext<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), x);
		CCE();
		ntt_2_16k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_16k<<<param.nttLen/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (param.nttLen == 32768) {
		ntt_1_32k_ext<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), x);
		CCE();
		ntt_2_32k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_32k<<<param.nttLen/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (param.nttLen == 65536) {
		ntt_1_64k_ext<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), x);
		CCE();
		ntt_2_64k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_64k<<<param.nttLen/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
}
void _nttw(uint64 *X, uint32 *x, int coeffwords, int relinIdx, int dev, hipStream_t st) {
	if (param.nttLen == 16384) {
		ntt_1_16k_ext_block<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), x, param.logRelin, relinIdx, coeffwords);
		CCE();
		ntt_2_16k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_16k<<<param.nttLen/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (param.nttLen == 32768) {
		ntt_1_32k_ext_block<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), x, param.logRelin,relinIdx, coeffwords);
		CCE();
		ntt_2_32k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_32k<<<param.nttLen/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (param.nttLen == 65536) {
		ntt_1_64k_ext_block<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), x, param.logRelin,relinIdx, coeffwords);
		CCE();
		ntt_2_64k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_64k<<<param.nttLen/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
}
// !!! x has length of param.nttLen
void _intt(uint32 *x, uint64 *X, int crtidx, int dev, hipStream_t st) {
	if (param.nttLen == 16384) {
		intt_1_16k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), X);
		CCE();
		ntt_2_16k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		intt_3_16k_modcrt<<<param.nttLen/512, 64, 0, st>>>(x, ptrNttSwap(dev), crtidx);
		CCE();
	}
	else if (param.nttLen == 32768) {
		intt_1_32k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), X);
		CCE();
		ntt_2_32k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		intt_3_32k_modcrt<<<param.nttLen/512, 64, 0, st>>>(x, ptrNttSwap(dev), crtidx);
		CCE();
	}
	else if (param.nttLen == 65536) {
		intt_1_64k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev), X);
		CCE();
		ntt_2_64k<<<param.nttLen/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		intt_3_64k_modcrt<<<param.nttLen/512, 64, 0, st>>>(x, ptrNttSwap(dev), crtidx);
		CCE();
	}
}
//// all crt polynomials
// ntt
void ntt(uint64 *X, uint32 *x, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		_ntt(X+i*param.nttLen, x+i*param.crtLen, dev, st);
}
void nttw(uint64 *X, uint32 *x, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	for (int i=0; i<param._numEvalKey(lvl); i++)
		_nttw(X+i*param.nttLen, x, param._wordsCoeff(lvl), i, dev, st);
}
// intt holding result
void inttHold(uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		_intt(ptrNttHold(dev)+i*param.nttLen, X+i*param.nttLen, i, dev, st);
}
// intt without barrett copy result, x has param.nttLen
void inttDoubleDeg(uint32 *x, uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		_intt(ptrNttHold(dev)+i*param.nttLen, X+i*param.nttLen, i, dev, st);
	CSC(hipMemcpyAsync(x, ptrNttHold(dev),
			param._numCrtPrime(lvl)*param.nttLen*sizeof(uint32), hipMemcpyDeviceToDevice, st));
}
// intt without barrett copy result, x has param.crtLen
void intt(uint32 *x, uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	for (int i=0; i<param._numCrtPrime(lvl); i++) {
		_intt(ptrNttHold(dev)+i*param.nttLen, X+i*param.nttLen, i, dev, st);
		CSC(hipMemcpyAsync(x+i*param.crtLen, ptrNttHold(dev)+i*param.nttLen,
			param.crtLen*sizeof(uint32), hipMemcpyDeviceToDevice, st));
	}
}
// intt with barrett, x has param.crtLen
void inttMod(uint32 *x, uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		_intt(ptrNttHold(dev)+i*param.nttLen, X+i*param.nttLen, i, dev, st);
	barrett(x, lvl, dev, st);
}
void nttMul(uint64 *z, uint64 *y, uint64 *x, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	ntt_mul<<<(param.nttLen+63)/64, 64, 0, st>>>(z, y, x, param._numCrtPrime(lvl), param.nttLen);
}
void nttMulNX1(uint64 *z, uint64 *x, uint64 *scalar, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	ntt_mul_nx1<<<(param.nttLen+63)/64, 64, 0, st>>>(z, x, scalar, param._numCrtPrime(lvl), param.nttLen);
}
void nttAdd(uint64 *z, uint64 *y, uint64 *x, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	ntt_add<<<(param.nttLen+63)/64, 64, 0, st>>>(z, x, y, param._numCrtPrime(lvl), param.nttLen);
}
void nttAddNX1(uint64 *z, uint64 *x, uint64 *scalar, int logq, int dev, hipStream_t st) {
	int lvl = param._getLevel(logq);
	ntt_add_nx1<<<(param.nttLen+63)/64, 64, 0, st>>>(z, x, scalar, param._numCrtPrime(lvl), param.nttLen);
}

void barrett(uint32 *dst, uint32 *src, int lvl, int dev, hipStream_t st) {
	hipSetDevice(dev);
	uint32 *ptrCrt = ptrBarrettCrt(dev);
	uint64 *ptrNtt = ptrBarrettNtt(dev);
	uint32 *ptrSrc = ptrBarrettSrc(dev);
	CSC(hipMemcpyAsync(ptrSrc, src, param._numCrtPrime(lvl)*param.nttLen*sizeof(uint32),
				hipMemcpyDeviceToDevice, st));
	// ptrSrc = f, deg = 2n-2
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		_ntt(ptrNtt+i*param.nttLen, ptrSrc+i*param.nttLen+param.modLen-1, dev, st);
	// ptrNtt = f>>(n-1), deg = n-1
	barrett_mul_un<<<(param.nttLen+63)/64, 64, 0, st>>>
			(ptrNtt, param._numCrtPrime(lvl), param.nttLen);
	inttDoubleDeg(ptrCrt, ptrNtt, param._logCoeff(lvl), dev, st);
	// ptrCrt = u * f>>(n-1), deg = 2n-2
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		CSC(hipMemsetAsync(ptrCrt+i*param.nttLen, 0, param.modLen*sizeof(uint32), st));
	// ptrCrt = u*f>>(2n-1)<<n
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		_ntt(ptrNtt+i*param.nttLen, ptrCrt+i*param.nttLen+param.modLen, dev, st);
	// ptrNtt = (u * f>>(n-1))>>n = u*f>>(2n-1), deg = n-2
	barrett_mul_mn<<<(param.nttLen+63)/64, 64, 0, st>>>
			(ptrNtt, param._numCrtPrime(lvl), param.nttLen);
	// ptrNtt = (m-x^n) * (u * f>>(n-1))>>n, deg = 2n-3
	barrett_sub_1<<<(param.modLen+63)/64, 64, 0, st>>>
			(ptrSrc, ptrCrt, param._numCrtPrime(lvl), param.modLen, param.nttLen);
	// ptrSrc = f - (u*f>>(2n-1))<<n
	inttDoubleDeg(ptrCrt, ptrNtt, param._logCoeff(lvl), dev, st);
	// ptrCrt = (m-x^n) * (u * f>>(2n-1)), deg = 2n-3
	barrett_sub_2<<<(param.nttLen+63)/64, 64, 0, st>>>
			(ptrSrc, ptrCrt, param._numCrtPrime(lvl), param.nttLen);
	// ptrSrc = f - (m*u*f)>>(2n-1), deg = n
	barrett_sub_mc<<<(param.nttLen+63)/64, 64, param._numCrtPrime(lvl)*sizeof(uint32), st>>>
			(ptrSrc, param._numCrtPrime(lvl), param.modLen, param.crtLen, param.nttLen);
	// ptrSrc = ptrSrc - m, deg = n-1
	for (int i=0; i<param._numCrtPrime(lvl); i++)
		CSC(hipMemcpyAsync(dst+i*param.crtLen, ptrSrc+i*param.nttLen,
				param.crtLen*sizeof(uint32), hipMemcpyDeviceToDevice, st));
}
void barrett(uint32 *dst, int lvl, int dev, hipStream_t st) {
	barrett(dst, inttResult(dev), lvl, dev, st);
}
} // end cuHE
